#include "cuda_utility.cuh"

int main()
{
	
	int number_of_steps = 1000;
	//Initialize data holder vectors
	std::vector<bool> reff_vec(h * w);
	std::vector<float> gpu_vec(h * w);

	//Creating a random number generator
	std::random_device rd{};
	std::mt19937 mersenne_engine{ rd() };
	std::uniform_real_distribution<float> dist{ 0, 100 };
	auto gen = [&dist, &mersenne_engine]() { return dist(mersenne_engine) < 50 ? false : true; };
	
	//Fill values of the 2 vector
	generate(reff_vec.begin(), reff_vec.end(), gen);
	std::copy(reff_vec.begin(), reff_vec.end(), gpu_vec.begin());
	
	//Creating the table object
	table t1(h, w, reff_vec);


	//Creating arrays for host and device
	float* host_array = gpu_vec.data();
	float* device_output;
	hipMalloc(&device_output, w * h * sizeof(float));

	//Output data
	std::ofstream handler_naive("data/naive_conway.txt");
	std::ofstream handler_gpu("data/texture_conway.txt");

	//Write out, the initial table
	write_out_result(host_array, handler_gpu);
	t1.write_table_out(handler_naive);

	//Executing the simulation
	for (int i = 0; i < number_of_steps; ++i)
	{
		//Creating the cuda texture object
		auto texObj = get_texobject(host_array);
		
		// One step 
		step(host_array,device_output);
		t1.do_game();

		//Write out results
		write_out_result(host_array, handler_gpu);
		t1.write_table_out(handler_naive);
	}

	
	//Free the allocated memory
	auto err = hipFree(device_output);
	if (err != hipSuccess) { std::cout << "Error freeing array allocation: " << hipGetErrorString(err) << "\n"; return -1; }
	
	handler_gpu.close();
	handler_naive.close();

}